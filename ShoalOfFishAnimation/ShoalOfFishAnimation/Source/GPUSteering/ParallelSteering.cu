#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono> 

#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <>

#include "../Vector/Vector.h"
#include "../Point/Point.h"
#include "ParallelSteering.cuh"
#include "../Constant/Costant.h"

#pragma once

#ifdef NVCC
#define CU_LAUNCH(...) <<<__VA_ARGS__>>>
#else
#define CU_LAUNCH(...) 
#define __launch_bounds__(...)
#define __syncwarp()
#define __syncthreads()
#define __any_sync() (0)
#define __all_sync() (0)
#define __ballot_sync() (0)
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__ struct CudaVector
{
public:
	float X, Y;
	__device__ CudaVector(float x, float y);
};
__device__ CudaVector::CudaVector(float x, float y)
{
	this->X = x;
	this->Y = y;
}

__global__ void MoveFish(Fish *fish, float mouseX, float mouseY)
{
	int threadIndex = threadIdx.x;
	int blockIndex = blockIdx.x;
	threadIndex = (blockIndex * 1024) + threadIndex;
	if (threadIndex < FISH_COUNT)
	{
		float fishPositionX = fish[threadIndex].Position.X;
		float fishPositionY = fish[threadIndex].Position.Y;

		CudaVector averageDirection = CudaVector(fish[threadIndex].Direction.X, fish[threadIndex].Direction.Y);
		#pragma region SteerToTheAverageHeadingOfLocalFlockmates
		if(!fish[threadIndex].Settings.independence)
		{
			for (int i = 0; i < FISH_COUNT; i++)
			{
				if (i == threadIndex) continue;
				float x = fish[i].Position.X;
				float y = fish[i].Position.Y;
				if ((fishPositionX - x)*(fishPositionX - x) + (fishPositionY - y)*(fishPositionY - y) < FISH_VIEW_RANGE*FISH_VIEW_RANGE)
				{
					//avarageDirection += fishes[i].Direction;
					averageDirection.X += fish[i].Direction.X;
					averageDirection.Y += fish[i].Direction.Y;
				}
			}
			float length = sqrt(averageDirection.X*averageDirection.X + averageDirection.Y * averageDirection.Y);
			if (length > 0.0001)
			{
				averageDirection.X /= length;
				averageDirection.Y /= length;
			}
			else
			{
				averageDirection.X = fish[threadIndex].Direction.X;
				averageDirection.Y = fish[threadIndex].Direction.Y;
			}
		}
		#pragma endregion
		CudaVector groupingDirection = CudaVector(0, 0);
		#pragma region SteerToTheAveragePositionOfLocalFlockmates
		if(fish[threadIndex].Settings.grouping)
		{
			CudaVector groupingPoint = CudaVector(fishPositionX, fishPositionY);
			int n = 1;
			for (int i = 0; i < FISH_COUNT; i++)
			{
				if (i == threadIndex) continue;
				float x = fish[i].Position.X;
				float y = fish[i].Position.Y;
				float dist_x = fishPositionX - x;
				float dist_y = fishPositionY - y;
				
				if (dist_x * dist_x + dist_y * dist_y < FISH_VIEW_RANGE*FISH_VIEW_RANGE)
				{
					groupingPoint.X += x;
					groupingPoint.Y += y;
					n++;
				}
			}
			groupingPoint.X /= n;
			groupingPoint.Y /= n;
			groupingDirection.X = (groupingPoint.X - fishPositionX);
			groupingDirection.Y = (groupingPoint.Y - fishPositionY);
			float length = sqrt(groupingDirection.X*groupingDirection.X + groupingDirection.Y * groupingDirection.Y);
			if (abs(length) > 0.001)
			{
				groupingDirection.X /= length;
				groupingDirection.Y /= length;
			}
		}
		#pragma endregion
		CudaVector antyCrowdingVector = CudaVector(0, 0);
		#pragma region SteerToAvoidCrowdingLocalFlockmates
		if(fish[threadIndex].Settings.grouping)
		{
			for (int i = 0; i < FISH_COUNT; i++)
			{
				float x = fish[i].Position.X;
				float y = fish[i].Position.Y;
				if (abs(x - fish[threadIndex].Position.X) < 0.001 && (y - fish[threadIndex].Position.Y) < 0.001) continue;
				//Vector fromLocalToNeigh = Vector(x - Position.X, y - Position.Y);
				float fromLocalToNeigh_X = x - fish[threadIndex].Position.X;
				float fromLocalToNeigh_Y = y - fish[threadIndex].Position.Y;
				float fromLocalToNeighDist = sqrt((fromLocalToNeigh_X*fromLocalToNeigh_X) + (fromLocalToNeigh_Y*fromLocalToNeigh_Y));
				if (fromLocalToNeighDist <= (FISH_COLISION_RANGE * fish[i].Settings.size + FISH_COLISION_RANGE * fish[threadIndex].Settings.size))
				{
					fromLocalToNeigh_X /= fromLocalToNeighDist;
					fromLocalToNeigh_Y /= fromLocalToNeighDist;
					fromLocalToNeigh_X *= ((FISH_COLISION_RANGE * fish[i].Settings.size + FISH_COLISION_RANGE * fish[threadIndex].Settings.size) - fromLocalToNeighDist);
					fromLocalToNeigh_Y *= ((FISH_COLISION_RANGE * fish[i].Settings.size + FISH_COLISION_RANGE * fish[threadIndex].Settings.size) - fromLocalToNeighDist);
					antyCrowdingVector.X -= fromLocalToNeigh_X;
					antyCrowdingVector.Y -= fromLocalToNeigh_Y;
				}
			}
		}
		#pragma endregion
		CudaVector avoidMouse = CudaVector(0, 0);
		#pragma region AvoidingMouse
		float dist = sqrt((mouseX - fishPositionX)*(mouseX - fishPositionX) + (mouseY - fishPositionY)*(mouseY - fishPositionY));
		if (dist <= MOUSE_FEAR_DISTANCE)
		{
			avoidMouse.X = ((fishPositionX - mouseX) / dist) * (MOUSE_FEAR_DISTANCE - dist);
			avoidMouse.Y = ((fishPositionY - mouseY) / dist) * (MOUSE_FEAR_DISTANCE - dist);
		}
		#pragma endregion

		

		__syncthreads();
		float x_move = groupingDirection.X + antyCrowdingVector.X * 0.3 + averageDirection.X * fish[threadIndex].Settings.speed + avoidMouse.X;
		float y_move = groupingDirection.Y + antyCrowdingVector.Y * 0.3 + averageDirection.Y * fish[threadIndex].Settings.speed + avoidMouse.Y;
		fish[threadIndex].Position.X += x_move;
		fish[threadIndex].Position.Y += y_move;

		#pragma region Window frame
		if (fish[threadIndex].Position.X > MATRIX_HALF_WIDTH)
		{
			fish[threadIndex].Position.X = -MATRIX_HALF_WIDTH;
			fish[threadIndex].Position.Y *= -1;
		}
		if (fish[threadIndex].Position.X < -MATRIX_HALF_WIDTH)
		{
			fish[threadIndex].Position.X = MATRIX_HALF_WIDTH;
			fish[threadIndex].Position.Y *= -1;
		}
		if (fish[threadIndex].Position.Y > MATRIX_HALF_HEIGHT)
		{
			fish[threadIndex].Position.X *= -1;
			fish[threadIndex].Position.Y = -MATRIX_HALF_HEIGHT;
		}
		if (fish[threadIndex].Position.Y < -MATRIX_HALF_HEIGHT)
		{
			fish[threadIndex].Position.X *= -1;
			fish[threadIndex].Position.Y = MATRIX_HALF_HEIGHT;
		}
		#pragma endregion
		
		float directionLength = sqrt((averageDirection.X + avoidMouse.X) * (averageDirection.X + avoidMouse.X) + (averageDirection.Y + avoidMouse.Y)*(averageDirection.Y + avoidMouse.Y));
		if (directionLength > 0.0001)
		{
			fish[threadIndex].Direction.X = (averageDirection.X + avoidMouse.X) / directionLength;
			fish[threadIndex].Direction.Y = (averageDirection.Y + avoidMouse.Y) / directionLength;
		}
		else
		{
			fish[threadIndex].Direction.X = averageDirection.X;
			fish[threadIndex].Direction.Y = averageDirection.Y;
		}
	}

}

Fish* d_fish;

extern "C" void InitParallerlSteering(Fish* h_fish)
{
	hipMalloc((void**)&d_fish, FISH_COUNT * sizeof(Fish));
	hipMemcpy(d_fish, h_fish, FISH_COUNT * sizeof(Fish), hipMemcpyHostToDevice);
}

extern "C" void FinalizeParallerlSteering()
{
	hipFree(d_fish);
}

extern "C" void ParallelSteering(Fish* h_fish, float MouseX, float MouseY)
{
	MoveFish << <1 + (FISH_COUNT/1024), 1024 >> > (d_fish, MouseX, MouseY);
	gpuErrchk(hipPeekAtLastError());
	hipDeviceSynchronize();
	hipMemcpy(h_fish, d_fish, FISH_COUNT * sizeof(Fish), hipMemcpyDeviceToHost);
}

