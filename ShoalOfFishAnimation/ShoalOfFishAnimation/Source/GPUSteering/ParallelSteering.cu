#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono> 

#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <>

#include "../Vector/Vector.h"
#include "../Point/Point.h"
#include "ParallelSteering.cuh"
#include "../Constant/Costant.h"

#pragma once

#ifdef NVCC
#define CU_LAUNCH(...) <<<__VA_ARGS__>>>
#else
#define CU_LAUNCH(...) 
#define __launch_bounds__(...)
#define __syncwarp()
#define __syncthreads()
#define __any_sync() (0)
#define __all_sync() (0)
#define __ballot_sync() (0)
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__ struct CudaVector
{
public:
	float X, Y;
	__device__ CudaVector(float x, float y);
};
__device__ CudaVector::CudaVector(float x, float y)
{
	this->X = x;
	this->Y = y;
}

__global__ void MoveFish(Fish *fish)
{
	int threadIndex = threadIdx.x;
	int blockIndex = blockIdx.x;
	threadIndex = (blockIndex * 1024) + threadIndex;
	if (threadIndex < FISH_COUNT)
	{
		float fishPositionX = fish[threadIndex].Position.X;
		float fishPositionY = fish[threadIndex].Position.Y;

		CudaVector averageDirection = CudaVector(fish[threadIndex].Direction.X, fish[threadIndex].Direction.Y);
		#pragma region SteerToTheAverageHeadingOfLocalFlockmates
		{
			for (int i = 0; i < FISH_COUNT; i++)
			{
				if (i == threadIndex) continue;
				float x = fish[i].Position.X;
				float y = fish[i].Position.Y;
				if ((fishPositionX - x)*(fishPositionX - x) + (fishPositionY - y)*(fishPositionY - y) < FISH_VIEW_RANGE*FISH_VIEW_RANGE)
				{
					//avarageDirection += fishes[i].Direction;
					averageDirection.X += fish[i].Direction.X;
					averageDirection.Y += fish[i].Direction.Y;
				}
			}
			float length = sqrt(averageDirection.X*averageDirection.X + averageDirection.Y * averageDirection.Y);
			if (length > 0.0001)
			{
				averageDirection.X /= length;
				averageDirection.Y /= length;
			}
			else
			{
				averageDirection.X = fish[threadIndex].Direction.X;
				averageDirection.Y = fish[threadIndex].Direction.Y;
			}
		}
		#pragma endregion
		CudaVector groupingDirection = CudaVector(0, 0);
		#pragma region SteerToTheAveragePositionOfLocalFlockmates
		{
			CudaVector groupingPoint = CudaVector(fishPositionX, fishPositionY);
			int n = 1;
			for (int i = 0; i < FISH_COUNT; i++)
			{
				if (i == threadIndex) continue;
				float x = fish[i].Position.X;
				float y = fish[i].Position.Y;
				float dist_x = fishPositionX - x;
				float dist_y = fishPositionY - y;
				
				if (dist_x * dist_x + dist_y * dist_y < FISH_VIEW_RANGE*FISH_VIEW_RANGE)
				{
					groupingPoint.X += x;
					groupingPoint.Y += y;
					n++;
				}
			}
			groupingPoint.X /= n;
			groupingPoint.Y /= n;
			groupingDirection.X = (groupingPoint.X - fishPositionX);
			groupingDirection.Y = (groupingPoint.Y - fishPositionY);
			float length = sqrt(groupingDirection.X*groupingDirection.X + groupingDirection.Y * groupingDirection.Y);
			if (abs(length) > 0.001)
			{
				groupingDirection.X /= length;
				groupingDirection.Y /= length;
			}
		}
		#pragma endregion
		CudaVector antyCrowdingVector = CudaVector(0, 0);
		#pragma region SteerToAvoidCrowdingLocalFlockmates
		{
			for (int i = 0; i < FISH_COUNT; i++)
			{
				float x = fish[i].Position.X;
				float y = fish[i].Position.Y;
				if (abs(x - fish[threadIndex].Position.X) < 0.001 && (y - fish[threadIndex].Position.Y) < 0.001) continue;
				//Vector fromLocalToNeigh = Vector(x - Position.X, y - Position.Y);
				float fromLocalToNeigh_X = x - fish[threadIndex].Position.X;
				float fromLocalToNeigh_Y = y - fish[threadIndex].Position.Y;
				float fromLocalToNeighDist = sqrt((fromLocalToNeigh_X*fromLocalToNeigh_X) + (fromLocalToNeigh_Y*fromLocalToNeigh_Y));
				if (fromLocalToNeighDist <= (FISH_COLISION_RANGE * 2))
				{
					fromLocalToNeigh_X /= fromLocalToNeighDist;
					fromLocalToNeigh_Y /= fromLocalToNeighDist;
					fromLocalToNeigh_X *= ((FISH_COLISION_RANGE * 2) - fromLocalToNeighDist);
					fromLocalToNeigh_Y *= ((FISH_COLISION_RANGE * 2) - fromLocalToNeighDist);
					antyCrowdingVector.X -= fromLocalToNeigh_X;
					antyCrowdingVector.Y -= fromLocalToNeigh_Y;
				}
			}
		}
		#pragma endregion

		__syncthreads();
		float x_move = groupingDirection.X + antyCrowdingVector.X * 0.3 + averageDirection.X * 2;
		float y_move = groupingDirection.Y + antyCrowdingVector.Y * 0.3 + averageDirection.Y * 2;
		fish[threadIndex].Position.X += x_move;
		fish[threadIndex].Position.Y += y_move;

		#pragma region Window frame
		if (fish[threadIndex].Position.X > MATRIX_HALF_WIDTH)
		{
			fish[threadIndex].Position.X = -MATRIX_HALF_WIDTH;
			fish[threadIndex].Position.Y *= -1;
		}
		if (fish[threadIndex].Position.X < -MATRIX_HALF_WIDTH)
		{
			fish[threadIndex].Position.X = MATRIX_HALF_WIDTH;
			fish[threadIndex].Position.Y *= -1;
		}
		if (fish[threadIndex].Position.Y > MATRIX_HALF_HEIGHT)
		{
			fish[threadIndex].Position.X *= -1;
			fish[threadIndex].Position.Y = -MATRIX_HALF_HEIGHT;
		}
		if (fish[threadIndex].Position.Y < -MATRIX_HALF_HEIGHT)
		{
			fish[threadIndex].Position.X *= -1;
			fish[threadIndex].Position.Y = MATRIX_HALF_HEIGHT;
		}
		#pragma endregion
		
		fish[threadIndex].Direction.X = averageDirection.X;
		fish[threadIndex].Direction.Y = averageDirection.Y;
	}

}

Fish* d_fish;

extern "C" void InitParallerlSteering(Fish* h_fish, int count)
{
	hipMalloc((void**)&d_fish, count * sizeof(Fish));
	hipMemcpy(d_fish, h_fish, count * sizeof(Fish), hipMemcpyHostToDevice);
}

extern "C" void FinalizeParallerlSteering()
{
	hipFree(d_fish);
}

extern "C" void ParallelSteering(Fish* h_fish, int count)
{
	MoveFish << <1 + (FISH_COUNT/1024), 1024 >> > (d_fish);
	gpuErrchk(hipPeekAtLastError());
	hipDeviceSynchronize();
	hipMemcpy(h_fish, d_fish, count * sizeof(Fish), hipMemcpyDeviceToHost);	
}

