#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono> 

#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <>
#include <string>
#include <sstream>

#include "../Vector/Vector.h"
#include "../Point/Point.h"
#include "ParallelSteering.cuh"
#include "../Constant/Constant.h"

#pragma once

#ifdef NVCC
#define CU_LAUNCH(...) <<<__VA_ARGS__>>>
#else
#define CU_LAUNCH(...) 
#define __launch_bounds__(...)
#define __syncwarp()
#define __syncthreads()
#define __any_sync() (0)
#define __all_sync() (0)
#define __ballot_sync() (0)
#endif

#pragma region Cuda error
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
#pragma endregion

#pragma region Cuda Vector
__device__ struct CudaVector
{
public:
	float X, Y;
	__device__ CudaVector(float x, float y);
};
__device__ CudaVector::CudaVector(float x, float y)
{
	this->X = x;
	this->Y = y;
}
#pragma endregion

#pragma region Kernel
__global__ void MoveFish(Fish *fish, float mouseX, float mouseY)
{
	int threadIndex = threadIdx.x;
	int blockIndex = blockIdx.x;
	threadIndex = (blockIndex * 1024) + threadIndex;
	if (threadIndex < FISH_COUNT)
	{
		float fishPositionX = fish[threadIndex].Position.X;
		float fishPositionY = fish[threadIndex].Position.Y;
		float fishSize = fish[threadIndex].Settings.size;

		CudaVector averageDirection = CudaVector(fish[threadIndex].Direction.X, fish[threadIndex].Direction.Y);
		CudaVector groupingDirection = CudaVector(0, 0);
		CudaVector antyCrowdingVector = CudaVector(0, 0);
		#pragma region Steering
		{
			CudaVector groupingPoint = CudaVector(fishPositionX, fishPositionY);
			int n = 1;
			for (int i = 0; i < FISH_COUNT; i++)
			{
				if (i == threadIndex) continue;
				float x = fish[i].Position.X;
				float y = fish[i].Position.Y;
				float distance_X = (fishPositionX - x);
				float distance_Y = (fishPositionY - y);
				float distance = sqrt(distance_X*distance_X+distance_Y*distance_Y);
				if ( distance < FISH_VIEW_RANGE)
				{
					#pragma region SteerToTheAverageHeadingOfLocalFlockmates
					if (!fish[threadIndex].Settings.independence)
					{
						averageDirection.X += fish[i].Direction.X;
						averageDirection.Y += fish[i].Direction.Y;
					}
					#pragma endregion

					#pragma region SteerToTheAveragePositionOfLocalFlockmates
					if (fish[threadIndex].Settings.grouping)
					{
						groupingPoint.X += x;
						groupingPoint.Y += y;
						n++;
					}
					#pragma endregion
				}
				#pragma region  SteerToAvoidCrowdingLocalFlockmates
				if (fish[threadIndex].Settings.grouping)
				{
					float colisionRange = (FISH_COLISION_RANGE * fish[i].Settings.size) + (FISH_COLISION_RANGE * fishSize);
					if (distance < colisionRange)
					{
						antyCrowdingVector.X += (((colisionRange*distance_X) / distance) - distance_X);
						antyCrowdingVector.Y += (((colisionRange*distance_Y) / distance) - distance_Y);
					}
				}
				#pragma endregion
			}
			#pragma region SteerToTheAverageHeadingOfLocalFlockmates
			float length = sqrt(averageDirection.X*averageDirection.X+averageDirection.Y*averageDirection.Y);
			if (!fish[threadIndex].Settings.independence)
			{
				if (length > 0.0001)
				{
					averageDirection.X /= length;
					averageDirection.Y /= length;
				}
				else
				{
					averageDirection.X = fish[threadIndex].Direction.X;
					averageDirection.Y = fish[threadIndex].Direction.Y;
				}
			}
			#pragma endregion

			#pragma region SteerToTheAveragePositionOfLocalFlockmates
			if (fish[threadIndex].Settings.grouping)
			{
				groupingPoint.X /= n;
				groupingPoint.Y /= n;
				groupingDirection.X = (groupingPoint.X - fishPositionX);
				groupingDirection.Y = (groupingPoint.Y - fishPositionY);
				length = sqrt(groupingDirection.X*groupingDirection.X + groupingDirection.Y*groupingDirection.Y);
				if (length > 0.001)
				{
					groupingDirection.X /= length;
					groupingDirection.Y /= length;
				}
			}
			#pragma endregion
		}
		#pragma endregion
		CudaVector avoidMouse = CudaVector(0, 0);
		#pragma region AvoidingMouse
		float dist = sqrt((mouseX - fishPositionX)*(mouseX - fishPositionX) + (mouseY - fishPositionY)*(mouseY - fishPositionY));
		if (dist <= MOUSE_FEAR_DISTANCE)
		{
			avoidMouse.X = ((fishPositionX - mouseX) / dist) * (MOUSE_FEAR_DISTANCE - dist);
			avoidMouse.Y = ((fishPositionY - mouseY) / dist) * (MOUSE_FEAR_DISTANCE - dist);
		}
		#pragma endregion



		__syncthreads();
		float x_move = groupingDirection.X + antyCrowdingVector.X * 0.3 + averageDirection.X * fish[threadIndex].Settings.speed + avoidMouse.X;
		float y_move = groupingDirection.Y + antyCrowdingVector.Y * 0.3 + averageDirection.Y * fish[threadIndex].Settings.speed + avoidMouse.Y;
		fish[threadIndex].Position.X += x_move;
		fish[threadIndex].Position.Y += y_move;

#pragma region Window frame
		if (fish[threadIndex].Position.X > MATRIX_HALF_WIDTH)
		{
			fish[threadIndex].Position.X = -MATRIX_HALF_WIDTH;
			fish[threadIndex].Position.Y *= -1;
		}
		if (fish[threadIndex].Position.X < -MATRIX_HALF_WIDTH)
		{
			fish[threadIndex].Position.X = MATRIX_HALF_WIDTH;
			fish[threadIndex].Position.Y *= -1;
		}
		if (fish[threadIndex].Position.Y > MATRIX_HALF_HEIGHT)
		{
			fish[threadIndex].Position.X *= -1;
			fish[threadIndex].Position.Y = -MATRIX_HALF_HEIGHT;
		}
		if (fish[threadIndex].Position.Y < -MATRIX_HALF_HEIGHT)
		{
			fish[threadIndex].Position.X *= -1;
			fish[threadIndex].Position.Y = MATRIX_HALF_HEIGHT;
		}
#pragma endregion

		float directionLength = sqrt((averageDirection.X + avoidMouse.X) * (averageDirection.X + avoidMouse.X) + (averageDirection.Y + avoidMouse.Y)*(averageDirection.Y + avoidMouse.Y));
		if (directionLength > 0.0001)
		{
			fish[threadIndex].Direction.X = (averageDirection.X + avoidMouse.X) / directionLength;
			fish[threadIndex].Direction.Y = (averageDirection.Y + avoidMouse.Y) / directionLength;
		}
		else
		{
			fish[threadIndex].Direction.X = averageDirection.X;
			fish[threadIndex].Direction.Y = averageDirection.Y;
		}
	}

}
#pragma endregion

extern "C"  std::string ParallelSteering(Fish* h_fish, float MouseX, float MouseY) 
{
	Fish* d_fish;
	double mallocTime = 0;
	double calcualtionTime = 0;

	auto start = std::chrono::high_resolution_clock::now();
	#pragma region HostToDevice
		hipMalloc((void**)&d_fish, FISH_COUNT * sizeof(Fish));
		hipMemcpy(d_fish, h_fish, FISH_COUNT * sizeof(Fish), hipMemcpyHostToDevice);
	#pragma endregion
	auto finish = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsedMem = (finish - start);

	start = std::chrono::high_resolution_clock::now();
	#pragma region Calcualtion
		MoveFish << <1 + (FISH_COUNT / 1024), 1024 >> > (d_fish, MouseX, MouseY);
		gpuErrchk(hipPeekAtLastError());
		hipDeviceSynchronize();
	#pragma endregion
	finish = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsedCal = (finish - start);

	start = std::chrono::high_resolution_clock::now();
	#pragma region DeviceToHost
		hipMemcpy(h_fish, d_fish, FISH_COUNT * sizeof(Fish), hipMemcpyDeviceToHost);
		hipFree(d_fish);
	#pragma endregion	
	finish = std::chrono::high_resolution_clock::now();
	elapsedMem += (finish - start);

	std::stringstream streams;
	streams << std::fixed << "| Calcualtion time : " << elapsedCal.count() << "s | Data copying time: " << elapsedMem.count() << "s ";

	return streams.str();
}

