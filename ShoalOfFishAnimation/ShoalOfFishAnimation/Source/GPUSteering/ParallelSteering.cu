#include "hip/hip_runtime.h"
#include <iostream>

#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <>

#include "../Vector/Vector.h"
#include "../Point/Point.h"
#include "ParallelSteering.cuh"

__device__ struct CudaVector
{
	float x, y;
	CudaVector(float x = 0, float y = 0);
	
	
};
__device__ CudaVector::CudaVector(float x, float y)
{
	this->x = x;
	this->y = y;
}

__global__ void MoveFish(Fish *fish, int count)
{
	int threadIndex = threadIdx.x;
	if (threadIndex < count)
	{
		/*Vector v = fish[threadIndex].VectorToTheAverageHeadingOfLocalFlockmates(fish, count);
		fish[threadIndex].MoveBy(v);*/
		//Takieo takie = Takieo(0, 2);
		fish[threadIndex].Position.X += fish[threadIndex].Direction.X;
		fish[threadIndex].Position.Y += fish[threadIndex].Direction.Y;
	}

}

extern "C" void ParallelSteering(Fish* h_fish, int count)
{
	Fish* d_fish;
	hipMalloc((void**)&d_fish, count * sizeof(Fish));
	hipMemcpy(d_fish, h_fish, count * sizeof(Fish), hipMemcpyHostToDevice);
	MoveFish << <1, count >> > (d_fish, count);
	hipDeviceSynchronize();
	hipMemcpy(h_fish, d_fish, count * sizeof(Fish), hipMemcpyDeviceToHost);
	hipFree(d_fish);
}

