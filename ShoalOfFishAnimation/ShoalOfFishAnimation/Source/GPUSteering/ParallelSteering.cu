#include "hip/hip_runtime.h"
#include <iostream>

#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <>

#include "../Vector/Vector.h"
#include "../Point/Point.h"
#include "ParallelSteering.cuh"
#include "../Constant/Costant.h"

#ifdef NVCC
#define CU_LAUNCH(...) <<<__VA_ARGS__>>>
#else
#define CU_LAUNCH(...) 
#define __launch_bounds__(...)
#define __syncwarp()
#define __syncthreads()
#define __any_sync() (0)
#define __all_sync() (0)
#define __ballot_sync() (0)
#endif

__global__ void MoveFish(Fish *fish)
{
	int threadIndex = threadIdx.x;
	if (threadIndex < FISH_COUNT)
	{
		#pragma region SteerToTheAverageHeadingOfLocalFlockmates

		#pragma endregion

		#pragma region SteerToTheAveragePositionOfLocalFlockmates
		float voctorToAvgPosition_X = 0, voctorToAvgPosition_Y = 0;
		{
			float avgPosition_X = fish[threadIndex].Position.X;
			float avgPosition_Y = fish[threadIndex].Position.Y;
			int n = 1;
			for (int i = 0; i < FISH_COUNT; i++)
			{
				if (i == threadIndex) continue;
				float x = fish[i].Position.X;
				float y = fish[i].Position.Y;
				float dist_x = fish[threadIndex].Position.X - x;
				float dist_y = fish[threadIndex].Position.Y - y;
				
				if (sqrt(dist_x * dist_x + dist_y * dist_y) < FISH_VIEW_RANGE)
				{
					avgPosition_X += x;
					avgPosition_Y += y;
					n++;
				}
			}
			avgPosition_X /= n;
			avgPosition_Y /= n;
			voctorToAvgPosition_X = (avgPosition_X - fish[threadIndex].Position.X);
			voctorToAvgPosition_Y = (avgPosition_Y - fish[threadIndex].Position.Y);
			float length = sqrt(voctorToAvgPosition_X*voctorToAvgPosition_X + voctorToAvgPosition_Y * voctorToAvgPosition_Y);
			if (abs(length) > 0.001)
			{
				voctorToAvgPosition_X /= length;
				voctorToAvgPosition_Y /= length;
			}
		}
		#pragma endregion
		
		#pragma region SteerToAvoidCrowdingLocalFlockmates
		float resultantVersor_X = 0, resultantVersor_Y = 0;
		for (int i = 0; i < FISH_COUNT; i++)
		{
			float x = fish[i].Position.X;
			float y = fish[i].Position.Y;
			if (abs(x - fish[threadIndex].Position.X) < 0.001 && (y - fish[threadIndex].Position.Y) < 0.001) continue;
			//Vector fromLocalToNeigh = Vector(x - Position.X, y - Position.Y);
			float fromLocalToNeigh_X = x - fish[threadIndex].Position.X;
			float fromLocalToNeigh_Y = y - fish[threadIndex].Position.Y;
			float fromLocalToNeighDist = sqrt((fromLocalToNeigh_X*fromLocalToNeigh_X) + (fromLocalToNeigh_Y*fromLocalToNeigh_Y));
			if (fromLocalToNeighDist <= (FISH_COLISION_RANGE * 2))
			{
				fromLocalToNeigh_X /= fromLocalToNeighDist;
				fromLocalToNeigh_Y /= fromLocalToNeighDist;
				fromLocalToNeigh_X *= ((FISH_COLISION_RANGE * 2) - fromLocalToNeighDist);
				fromLocalToNeigh_Y *= ((FISH_COLISION_RANGE * 2) - fromLocalToNeighDist);
				resultantVersor_X -= fromLocalToNeigh_X;
				resultantVersor_Y -= fromLocalToNeigh_Y;
			}
		}
		#pragma endregion

		__syncthreads();
		fish[threadIndex].Position.X += voctorToAvgPosition_X;		
		fish[threadIndex].Position.Y += voctorToAvgPosition_Y;
		fish[threadIndex].Position.X += resultantVersor_X*0.6;
		fish[threadIndex].Position.Y += resultantVersor_Y*0.6;
	}

}

__global__ void FishM(float* x, float* y)
{
	int threadIndex = threadIdx.x;
	if (threadIndex < FISH_COUNT)
	{
		float a_x = x[threadIndex];
		float a_y = y[threadIndex];
		int n = 1;
		for (int i = 0; i < FISH_COUNT; i++)
		{
			if (i == threadIndex) continue;
			float d_x = x[i] - x[threadIndex];
			float d_y = y[i] - y[threadIndex];
			if ((d_x*d_x + d_y * d_y) < FISH_VIEW_RANGE*FISH_VIEW_RANGE)
			{
				a_x += x[i];
				a_y += y[i];
				n++;
			}
		}
		a_x = a_x / n;
		a_y = a_y / n;
		x[threadIndex] = a_x;
		y[threadIndex] = a_y;
	}
}

extern "C" void ParallelSteering(Fish* h_fish, int count)
{
	/*float h_PosX[FISH_COUNT];
	float h_PosY[FISH_COUNT];
	for (int i = 0; i < FISH_COUNT; i++)
	{
		h_PosX[i] = h_fish[i].Position.X;
		h_PosY[i] = h_fish[i].Position.Y;
	}
	float* d_PosX;
	float* d_PosY;
	hipMalloc((void**)&d_PosX, FISH_COUNT * sizeof(float));
	hipMalloc((void**)&d_PosY, FISH_COUNT * sizeof(float));
	hipMemcpy(d_PosX, h_PosX, FISH_COUNT * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_PosY, h_PosY, FISH_COUNT * sizeof(float), hipMemcpyHostToDevice);
	FishM << <1, FISH_COUNT >> > (d_PosX, d_PosY);
	hipMemcpy(h_PosX, d_PosX, FISH_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_PosY, d_PosY, FISH_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_PosX);
	hipFree(d_PosY);
	for (int i = 0; i < FISH_COUNT; i++)
	{
		//h_fish[i].Position.X += h_PosX[i];
		//h_fish[i].Position.Y += h_PosY[i];
		printf("%d:po: %f - %f\n",i, h_PosX[i], h_PosY[i]);

	}*/
	Fish* d_fish;
	hipMalloc((void**)&d_fish, count * sizeof(Fish));
	hipMemcpy(d_fish, h_fish, count * sizeof(Fish), hipMemcpyHostToDevice);
	MoveFish << <1, count >> > (d_fish);
	hipDeviceSynchronize();
	hipMemcpy(h_fish, d_fish, count * sizeof(Fish), hipMemcpyDeviceToHost);
	hipFree(d_fish);
}

